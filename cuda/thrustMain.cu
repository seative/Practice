#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <vector>
#include <chrono>
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

template <typename T>
struct CudaAllocator
{
    using value_type = T;

    T *allocate(size_t size)
    {
        T *ptr = nullptr;
        hipMallocManaged(&ptr, size * sizeof(T));
        return ptr;
    }

    void deallocate(T *ptr, size_t size = 0)
    {
        hipFree(ptr);
    }

    template <typename... Args>
    void construct(T *p, Args &&...args)
    {
        if constexpr (!(sizeof...(Args) == 0 && std::is_pod_v<T>))
            ::new ((void *)p) T(std::forward<Args>(args)...);
    }
};

template <int N, class T>
__global__ void kernel(T *arr)
{
    // int i = blockDim.x * blockIdx.x + threadIdx.x;
    // if (i > n)
    //     return;
    // arr[i] = i;
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    {
        arr[i] = i;
    }
}

template <class Func>
__global__ void parallel_for(int n, Func func)
{
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
    {
        func(i);
    }
}

struct MyFunctor
{
    __device__ void operator()(int i) const
    {
        printf("number %d\n", i);
    }
};
__global__ void parallel_sum(int *sum, int const *arr, int n)
{
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
    {
        // sum[0] += arr[i];
        atomicAdd(&sum[0], arr[i]);
    }
}

int main()
{
    int n = 65536;
    std::vector<int, CudaAllocator<int>> arr(n);
    std::vector<int, CudaAllocator<int>> sum(1);
    // thrust::device_vector<int> arr(n);
    // thrust::device_vector<int> sum(1);

    for (int i = 0; i < n; ++i)
    {
        arr[i] = std::rand() % 4;
    }

    std::chrono::time_point tick1 = std::chrono::system_clock::now();
    parallel_sum<<<n / 128, 128>>>(sum.data(), arr.data(), n);
    hipDeviceSynchronize();
    std::chrono::time_point tick2 = std::chrono::system_clock::now();
    printf("result:%d\n", sum[0]);
    std::cout << std::chrono::duration_cast<std::chrono::microseconds>(tick2 - tick1).count() << std::endl;
    // float a = 3.14f;
    // thrust::host_vector<float> x_host(n);
    // thrust::device_vector<float> x_dev(n);

    // thrust::for_each(x_host.begin(), x_host.end(), [](float &x)
    //                  { x = std::rand() * (1.f / RAND_MAX); });
    // thrust::for_each(x_dev.begin(), x_dev.end(), [] __device__(float &x)
    //                  { x += 100.f; });

    // thrust::for_each(thrust::make_counting_iterator(0), thrust::make_counting_iterator(10), [] __device__(int i)
    //                  { printf("%d", i); });
    // for (int i = 0; i < n; ++i)
    // {
    //     x_host[i] = std::rand() * (1.f / RAND_MAX);
    //     y_host[i] = std::rand() * (1.f / RAND_MAX);
    // }

    // thrust::device_vector<float> x_dev = x_host;
    // thrust::device_vector<float> y_dev = y_host;

    // parallel_for<<<n / 512, 128>>>(n, [a, x = x_dev.data(), y = y_dev.data()] __device__(int i)
    //                                { x[i] = a * x[i] + y[i]; });

    // x_host = x_dev;

    // for (int i = 0; i < n; ++i)
    // {
    //     printf("x[%d] =%f\n", i, x_host[i]);
    // }
    // hipDeviceSynchronize();
    return 0;
}